#include "hip/hip_runtime.h"
#include <math.h>

#include <stdio.h>
#include <time.h>
#include <vector>

#include <SDL.h>
#include <pthread.h>

#include "bitmap.hh"
#include "gui.hh"
#include "util.hh"
#include "scheduler.hh"

using namespace std;

// Screen size
#define BMP_WIDTH 800
#define BMP_HEIGHT 600

// Cell dimension
#define CELL_DIM 5

// Grid size
#define GRID_WIDTH (BMP_WIDTH/CELL_DIM)
#define GRID_HEIGHT (BMP_HEIGHT/CELL_DIM)

// Threads per block
#define THREADS_PER_BLOCK 64

// Colors!
#define WHITE rgb32(255.,255.,255.)
#define BLACK rgb32(0.,0.,0.)

// barrier for threads
static pthread_barrier_t barrier;

// grid struct
typedef struct grid {
    bool board[(int) GRID_HEIGHT][(int) GRID_WIDTH];
} grid_t;

// coordinate struct
typedef struct coord {
    int x;
    int y;
} coord_t;

// for old times' sake
bool you_failed = true;

// mouse function parameter struct
typedef struct mouse_args {
    coord_t loc;
    uint32_t mouse_state;
    bool mouse_up;
    SDL_Event* event;
} mouse_args_t;

// keyboard function parameter struct
typedef struct keyboard_args {
    coord_t loc;
    const uint8_t* keyboard_state;
    SDL_Event* event;
} keyboard_args_t;

bool paused = true;

// bitmap screen variable
bitmap* bmp;

// grid for indicating cell state (dead or alive)
grid_t* g;

// Create a GUI window
gui ui("Conway's Game of Life", BMP_WIDTH, BMP_HEIGHT);


// Get input from the keyboard and execute proper command 
void* getKeyboardInput(void* params);

// Get input from the mouse and toggle the appropriate cell's state/color
void* getMouseInput(void* params);

// Update each cell in order to advance the simulation
void updateCells(void* params);

// display the screen 
void displayBMP(void* params);

// Toggle the cell's state, change the color accordingly
void toggleCell(coord_t loc);

// Set up the grid with an existing layout specified by a file
void loadGrid(FILE * layout);

//  Use Conway's update algorithm to decide whether or not to toggle cell 
__global__ void life_or_death(grid_t* gpu_g) {

    size_t index = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;

    // first thing's first: establish boundaries
    int row = index / GRID_WIDTH;
    int col = index % GRID_WIDTH;
    int left = max(0, col - 1);
    int right = min(GRID_WIDTH - 1, col + 1);
    int top = max(0, row - 1);
    int bottom = min(GRID_HEIGHT - 1, row + 1);

    int i,j;
    int alive_neighbors = 0;
    for(j = left; j <= right; j++) {
        for(i = top; i <= bottom; i++) {
            alive_neighbors += (!(col == j && row == i) && gpu_g->board[i][j]) ? 1 : 0;
        }
    }

    if (alive_neighbors < 2 || alive_neighbors > 3) {
        // clear the cell
        gpu_g->board[row][col] = false;
    }
    else if (alive_neighbors == 2) {
        // do nothing!
    }
    else { // (alive_neighbors == 3)
        // light up the cell
        gpu_g->board[row][col] = true;
    }
}

// Get input from the keyboard and execute proper command 
// UPDATE: currently, this only works as CTRL-P, CTRL-C, and CTRL-Q
void* getKeyboardInput(void* params) {

    bool clear = false;
    bool pause = false;
    bool quit = false;

    keyboard_args_t* args = (keyboard_args_t*) params;
    while (you_failed) {
        pthread_barrier_wait(&barrier); // Waits for the Poll Event in main

        // If the "c" key is pressed, clear the board
        switch (args->event->type) {
            case SDL_KEYDOWN:
                switch (args->event->key.keysym.scancode) {
                    case SDL_SCANCODE_C:
                        clear = true;
                        break;
                    case SDL_SCANCODE_P:
                        pause = true;
                        break;
                    case SDL_SCANCODE_Q:
                        quit = true;
                        break;
                    default:
                        break;
                }
                break;
            case SDL_KEYUP:
                switch (args->event->key.keysym.scancode) {
                    case SDL_SCANCODE_C:
                        if (clear) {
                            for (int x = 0; x < BMP_WIDTH; x++) {
                                for (int y = 0; y < BMP_HEIGHT; y++) {
                                    bmp->set(x, y, BLACK);
                                }
                            }
                            memset(g->board, 0, sizeof(grid_t));
                            puts("Cleared");
                            clear = false;
                        }
                        break;
                    case SDL_SCANCODE_P:
                        if (pause) {
                            paused = !(paused);
                            pause = false;
                            puts("Pause toggle!");
                        }
                        break;
                    case SDL_SCANCODE_Q:
                        if (quit) {
                            you_failed = false;
                            quit = false;
                        }
                        break;
                    default:
                        break;
                }
                break;
            default:
                break;
        }

        pthread_barrier_wait(&barrier); // Releases the main function to run updates
    }

    return NULL;
}


// Get input from the mouse and toggle the appropriate cell's state/color
void* getMouseInput(void* params) {

    mouse_args_t* args = (mouse_args_t*) params;
    while(you_failed) {
        pthread_barrier_wait(&barrier); // Waits for the Poll Event in main

        // If the left mouse button is pressed, get position and toggle cell
        // TO DO: make this thing toggle only once per click/release
        args->mouse_state = SDL_GetMouseState(&(args->loc.x), &(args->loc.y));

        if (args->mouse_state & SDL_BUTTON(SDL_BUTTON_LEFT)) {
            // Only create one if the mouse button has been released
            toggleCell(args->loc);
            if(args->mouse_up) {
                // Don't create another one until the mouse button is released
                args->mouse_up = false;
            }
        } 
        else {
            // The mouse button was released
            args->mouse_up = true;
        }

        pthread_barrier_wait(&barrier); // Releases the main function to run updates
    }

    return NULL;
}



// Update each cell in order to advance the simulation
void updateCells(void* params) {
    // allocate space for GPU grid
    grid_t* gpu_g;
    if (hipMalloc(&gpu_g, sizeof(grid_t)) != hipSuccess) {
        fprintf(stderr, "Failed to allocate grid on GPU\n");
        exit(2);
    }

    // alocate space for GPU bitmap
    bitmap* gpu_bmp;
    if (hipMalloc(&gpu_bmp, sizeof(bitmap)) != hipSuccess) {
        fprintf(stderr, "Failed to allocate bitmap on GPU\n");
        exit(2);
    }

    // copy the CPU grid to the GPU grid
    if (hipMemcpy(gpu_g, g, sizeof(grid_t), hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Failed to copy grid to the GPU\n");
    }

    // copy the CPU bitmap to the GPU bitmap
    if (hipMemcpy(gpu_bmp, bmp, sizeof(bitmap), hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Failed to copy bitmap to the GPU\n");
    }

    // number of block to run (rounding up to include all threads)
    size_t grid_blocks = (GRID_WIDTH*GRID_HEIGHT + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    life_or_death<<<grid_blocks, THREADS_PER_BLOCK>>>(gpu_g);
    hipDeviceSynchronize();

    // copy the GPU grid back to the CPU
    if (hipMemcpy(g, gpu_g, sizeof(grid_t), hipMemcpyDeviceToHost) != hipSuccess) {
        fprintf(stderr, "Failed to copy grid from the GPU\n");
    }

    // copy the CPU bitmap to the GPU bitmap
    if (hipMemcpy(bmp, gpu_bmp, sizeof(bitmap), hipMemcpyDeviceToHost) != hipSuccess) {
        fprintf(stderr, "Failed to copy bitmap from the GPU\n");
    }

    // Loop over points in the bitmap to change color
    for(int row = 0; row < BMP_HEIGHT; row++){
        for(int col = 0; col < BMP_WIDTH; col++){
            rgb32 color = g->board[row / CELL_DIM][col / CELL_DIM] ? WHITE : BLACK;
            bmp->set(col, row, color);
        }
    }

    // free everything we malloc'ed
    hipFree(gpu_g);
    hipFree(gpu_bmp);
}

// Toggle cell's color 
void toggleCell(coord_t loc) {

    // Indicate in the boolean grid that cell's state has been changed
    // color for cell to be set
    g->board[loc.y/CELL_DIM][loc.x/CELL_DIM] = true;
    rgb32 color = g->board[loc.y/CELL_DIM][loc.x/CELL_DIM] ? WHITE : BLACK;

    // Find upper-left corner in boolean grid of cell
    int x_start = (loc.x / CELL_DIM) * CELL_DIM;
    int y_start = (loc.y / CELL_DIM) * CELL_DIM;

    // Loop over points in the bitmap to change color
    for (int x = x_start; x < x_start + CELL_DIM; x++) {
        for (int y = y_start; y < y_start + CELL_DIM; y++) {
            bmp->set(x, y, color);
        }
    }
}

void loadGrid(FILE * layout) {
    coord_t loc;
    loc.x = 0, loc.y = 0;
    char ch;

    while ((ch = getc(layout)) != EOF) {
        if (ch == '\n') {
            loc.x = 0;
            loc.y ++;
        }
        else {
            if (ch != ' ') {
                toggleCell(loc);
            }
            loc.x ++;
        }
    }
}

/**
 * Entry point for the program
 */
int main(int argc, char ** argv) {

    // Create the bitmap 
    bitmap bits(BMP_WIDTH, BMP_HEIGHT);
    bmp = &bits;

    // Create the grid
    g = (grid_t*) malloc(sizeof(grid_t));
    for (int col = 0; col < GRID_WIDTH; col++) {
        for (int row = 0; row < GRID_HEIGHT; row++) {
            g->board[row][col] = false;
        }
    }

    if (argc > 1) {
        FILE * fp;
        fp = fopen(argv[1], "r");
        loadGrid(fp);
        fclose(fp);
    }

    SDL_Event event;

    // struct of arguments for mouse function
    mouse_args_t* mouse_args = (mouse_args_t*) malloc(sizeof(mouse_args_t));
    mouse_args->mouse_state = SDL_GetMouseState(&(mouse_args->loc.x), &(mouse_args->loc.y));
    mouse_args->mouse_up = true;
    mouse_args->event = &event;

    // struct of arguments for keyboard function
    keyboard_args_t* keyboard_args = (keyboard_args_t*) malloc(sizeof(keyboard_args_t));
    keyboard_args->keyboard_state = SDL_GetKeyboardState(NULL);
    keyboard_args->event = &event;

    ui.display(*bmp);

    // Initialize barrier
    pthread_barrier_init(&barrier, NULL, 3);

    // Set up threads
    pthread_t mouse_thread, keyboard_thread;

    if (pthread_create(&mouse_thread, NULL, getMouseInput, (void*) mouse_args)) {
        perror("error in pthread_create.\n");
        exit(2);
    }
    if (pthread_create(&keyboard_thread, NULL, getKeyboardInput, (void*) keyboard_args)) {
        perror("error in pthread_create.\n");
        exit(2);
    }

    // Loop until we get a quit event
    while(you_failed) {
        // Process events

        while(SDL_PollEvent(&event) == 1) {
            // If the event is a quit event, then leave the loop
            if(event.type == SDL_QUIT) {
                you_failed = false;
            }
        }

        // thread barrier for input threads
        pthread_barrier_wait(&barrier); // Releases the input threads to get input;
        pthread_barrier_wait(&barrier); // Waits for the input threads to finish

        if (!paused) {
            updateCells(keyboard_args);
            sleep_ms(25);
        }

        // Display the rendered frame
        ui.display(*bmp);
    }

    // join threads
    if (pthread_join(mouse_thread, NULL)) {
        perror("Failed joining.\n");
        exit(2);
    }
    if (pthread_join(keyboard_thread, NULL)) {
        perror("Failed joining.\n");
        exit(2);    
    }

    return 0;
}
